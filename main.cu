#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define N 100

__global__ void fun() {
    printf("%d\n", threadldx.x *
    threadldx.x);
}

int main() {
    fun<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}